#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "svmTrain.h"
#include "parse.hpp"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cblas.h>
#include <vector>
#include <string.h>
#include <getopt.h>
#include <math.h>
#include <vector>
#include "CycleTimer.h"

#include <thrust/host_vector.h> 
#include <thrust/device_vector.h> 
#include <thrust/copy.h> 
#include <thrust/fill.h> 
#include <thrust/sequence.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/inner_product.h>
#include <thrust/extrema.h>

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

using namespace std;

void SvmTrain::setup() {

	cout << "A late goodbye";
}

float clip_value(float num, float low, float high);
float rbf_kernel(thrust::host_vector<float> &x, int i1, int i2);
void get_x(float* x, float* x_copy, int idx, int num_attributes);
float get_train_accuracy(thrust::host_vector<float> &x, thrust::host_vector<int> &y, thrust::device_vector<float> &g_alpha, float b);

typedef struct {

	int num_attributes;
	int num_train_data;
	float c;
	float gamma;
	float epsilon;
	char input_file_name[60];
	char model_file_name[60];
	int max_iter;

} state_model;

//global structure for training parameters
static state_model state;

static void usage_exit() {
    cerr <<
"   Command Line:\n"
"\n"
"   -a/--num-att        :  [REQUIRED] The number of attributes\n"
"									  /features\n"
"   -x/--num-ex       	:  [REQUIRED] The number of training \n"
"									  examples\n"
"   -f/--file-path      :  [REQUIRED] Path to the training file\n"
"   -c/--cost        	:  Parameter c of the SVM (default 1)\n"
"   -g/--gamma       	:  Parameter gamma of the radial basis\n"
"						   function: exp(-gamma*|u-v|^2)\n"
"						   (default: 1/num-att)"
"   -e/--epsilon        :  Tolerance of termination criterion\n"
"						   (default 0.001)"
"	-n/--max-iter		:  Maximum number of iterations\n"
"						   (default 150,000"
"	-m/--model 			:  [REQUIRED] Path of model to be saved\n"
"\n";
    
	exit(-1);
}

static struct option longOptionsG[] =
{
    { "num-att",        required_argument,          0,  'a' },
    { "num-ex",         required_argument,          0,  'x' },
    { "cost",           required_argument,          0,  'c' },
    { "gamma",          required_argument,          0,  'g' },
    { "file-path",      required_argument,          0,  'f' },
    { "epsilon",       	required_argument,          0,  'e' },
    { "max-iter",		required_argument,			0,	'n'	},
    { "model",			required_argument,			0,	'm' },
    { 0,                0,                          0,   0  }
};

static void parse_arguments(int argc, char* argv[]) {

    // Default Values
    state.epsilon = 0.001;
    state.c = 1;
	state.num_attributes = -1;
	state.num_train_data = -1;
	state.gamma = -1;
	strcpy(state.input_file_name, "");
	strcpy(state.model_file_name, "");
	state.max_iter = 150000;

    // Parse args
    while (1) {
        int idx = 0;
        int c = getopt_long(argc, argv, "a:x:c:g:f:e:n:m:", longOptionsG, &idx);

        if (c == -1) {
            // End of options
            break;
        }

        switch (c) {
        case 'a':
            state.num_attributes = atoi(optarg);
            break;
        case 'x':
            state.num_train_data = atoi(optarg);
            break;
        case 'c':
            state.c = atof(optarg);
            break;
        case 'g':
            state.gamma = atof(optarg);
            break;
       case 'f':
            strcpy(state.input_file_name, optarg);
            break;
       case 'e':
            state.epsilon = atof(optarg);
            break;
       case 'n':
       		state.max_iter = atoi(optarg);
       		break;
       case 'm':
       		strcpy(state.model_file_name, optarg);
       		break;
        default:
            cerr << "\nERROR: Unknown option: -" << c << "\n";
            // Usage exit
            usage_exit();
        }
    }

	if(strcmp(state.input_file_name,"")==0 || strcmp(state.model_file_name,"")==0) {

		cerr << "Enter a valid file name\n";
		usage_exit();
	}

	if(state.num_attributes <= 0 || state.num_train_data <= 0) {

		cerr << "Missing a required parameter, or invalid parameter\n";
		usage_exit();

	}

	if(state.gamma < 0) {

		state.gamma = 1 / state.num_attributes;
	}

}

// Scalars
const float alpha = 1;
const float beta = 0;


struct arbitrary_functor
{

	const float C; 

	arbitrary_functor(float _c) : C(_c) {}

    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
        // I_set[i] = Alpha[i],  Y[i] , f[i], I_set1[i], I_set2[i];
		if(thrust::get<0>(t) == 0) {
		
			if(thrust::get<1>(t) == 1) {
			
				
				thrust::get<3>(t) = thrust::get<2>(t);
				
			}
			
			else {
				
				thrust::get<4>(t) = thrust::get<2>(t);
				
			}

		}	else if(thrust::get<0>(t) == C) {
		
			if(thrust::get<1>(t) == -1) {
			
				thrust::get<3>(t) = thrust::get<2>(t);
				
			}
			
			else {
				
				thrust::get<4>(t) = thrust::get<2>(t);
				
			}

		}	else {
		
			thrust::get<3>(t) = thrust::get<2>(t);
			thrust::get<4>(t) = thrust::get<2>(t);
			
		}
	}
};


struct update_functor
{
	const float gamma;
	const float alpha_lo_old;
	const float alpha_hi_old;
	const float alpha_lo_new;
	const float alpha_hi_new;
	const int y_lo;
	const int y_hi;
	const float x_hi_sq;
	const float x_lo_sq;

	update_functor(float _gamma, float _alpha_lo_old, float _alpha_hi_old, float _alpha_lo_new, float _alpha_hi_new, int _y_lo, int _y_hi, float _x_hi_sq, float _x_lo_sq) : 

	gamma(_gamma), 
	alpha_lo_old(_alpha_lo_old), 
	alpha_hi_old(_alpha_hi_old), 
	alpha_lo_new(_alpha_lo_new), 
	alpha_hi_new(_alpha_hi_new), 
	y_lo(_y_lo), 
	y_hi(_y_hi),
	x_hi_sq(_x_hi_sq),
	x_lo_sq(_x_lo_sq) 

	{}

    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
		float rbf_hi = expf(-1 * gamma * (thrust::get<2>(t) + x_hi_sq - (2*thrust::get<0>(t)) ));
		//printf("%f\t%f\n" , -1 * gamma * (thrust::get<2>(t) + x_hi_sq - (2*thrust::get<0>(t)) ) , rbf_hi);
		float rbf_lo = expf(-1 * gamma * (thrust::get<2>(t) + x_lo_sq - (2*thrust::get<1>(t)) ));
		//printf("%f\t%f\n" , -1 * gamma * (thrust::get<2>(t) + x_lo_sq - (2*thrust::get<1>(t)) ) , rbf_lo);

		float delta = (((alpha_hi_new-alpha_hi_old)*y_hi*rbf_hi) + ((alpha_lo_new - alpha_lo_old)*y_lo*rbf_lo));
	
		thrust::get<3>(t) += delta;	
	}
};


static hipblasHandle_t handle;
static hipStream_t stream1;
static hipStream_t stream2;

//static float* raw_g_hi_dotprod; 
//static float* raw_g_lo_dotprod; 

thrust::device_vector<float>& get_g_hi_dp() {

	static thrust::device_vector<float> g_hi_dotprod (state.num_train_data);
	return g_hi_dotprod;
}


thrust::device_vector<float>& get_g_lo_dp() {

	static thrust::device_vector<float> g_lo_dotprod (state.num_train_data);
	return g_lo_dotprod;

}

//static thrust::device_vector<float> g_lo_dotprod (state.num_train_data);

int prev_hi;
int prev_lo;

myCache* lineCache;	

thrust::device_vector<float>& lookup_cache(int I_idx, bool& cache_hit) {

	//static thrust::device_vector<float> g_hi_dotprod (state.num_train_data);
	thrust::device_vector<float>* lookup = lineCache->lookup(I_idx);
	if(lookup != NULL){
		cache_hit = true;
		return *lookup;
	}

	else {
		cache_hit = false;
		return lineCache->get_new_cache_line(I_idx);

	}
}

//Allocate x_hi, x_lo and an empty vector in device	i
void init_cuda_handles() {

	hipblasStatus_t status;
	hipError_t cudaStat;
	
	status = hipblasCreate(&handle);
	
	if (status != HIPBLAS_STATUS_SUCCESS) { 

		cout << "CUBLAS initialization failed\n"; 
		exit(EXIT_FAILURE); 
	}

	cudaStat = hipStreamCreate(&stream1);
	cudaStat = hipStreamCreate(&stream2);

	if (cudaStat == hipErrorInvalidValue) { 

		cout << "CUDA stream initialization failed\n"; 
		exit(EXIT_FAILURE); 
	}

//	thrust::device_vector<float>& g_lo_dotprod  = get_g_lo_dp();
//	thrust::device_vector<float>& g_hi_dotprod  = get_g_hi_dp();

//	raw_g_hi_dotprod = thrust::raw_pointer_cast(&g_hi_dotprod[0]);
//	raw_g_lo_dotprod = thrust::raw_pointer_cast(&g_lo_dotprod[0]);

//	prev_hi = -1;
//	prev_lo = -1;
	
}

void destroy_cuda_handles() {

	hipblasDestroy(handle);

}


inline int update_f(thrust::device_vector<float> &g_f, float* raw_g_x, thrust::device_vector<float> g_x_sq, int I_lo, int I_hi, int y_lo, int y_hi, float alpha_lo_old, float alpha_hi_old, float alpha_lo_new, float alpha_hi_new) {

//	unsigned long long t1,t2;
	
//	t1 = CycleTimer::currentTicks();
	
	cout << I_hi << "," << I_lo << "\n";

	bool hi_hit;
	bool lo_hit;

	thrust::device_vector<float>& g_hi_dotprod  = lookup_cache(I_hi, hi_hit);
	thrust::device_vector<float>& g_lo_dotprod  = lookup_cache(I_lo, lo_hit);
	
	float* raw_g_hi_dotprod = thrust::raw_pointer_cast(&g_hi_dotprod[0]);
	float* raw_g_lo_dotprod = thrust::raw_pointer_cast(&g_lo_dotprod[0]);

	printf("%x, %x\n",raw_g_hi_dotprod, raw_g_lo_dotprod);

	//cout << "UPDATE_F: " << t2-t1 << "\n";
	//t1 = t2;

	//thrust::device_vector<float> g_hi_dotprod (state.num_train_data);

	if(!hi_hit) {

		cout << "HI MISS\n";

		hipblasSetStream(handle, stream1);

//	t2 = CycleTimer::currentTicks();
//	cout << "UPDATE_F, INIT: " << t2-t1 << "\n";
//	t1 = t2;
		
		hipblasSgemv( handle, HIPBLAS_OP_T, state.num_attributes, state.num_train_data, &alpha, raw_g_x, state.num_attributes, &raw_g_x[I_hi * state.num_attributes], 1, &beta, raw_g_hi_dotprod, 1 );
	
//	t2 = CycleTimer::currentTicks();
//	cout << "SGEMV 1: " << t2-t1 << "\n";
//	t1 = t2;
	}

	cout << "----------------\n";

	for (int i = 0 ; i < state.num_attributes; i++) {

		cout << g_hi_dotprod[i] << ",";

	}

	cout << "\n-------------\n";
	
	if(!lo_hit) {

		cout << "LO MISS \n";

		hipblasSetStream(handle, stream2);
	
		hipblasSgemv( handle, HIPBLAS_OP_T, state.num_attributes, state.num_train_data, &alpha, raw_g_x, state.num_attributes, &raw_g_x[I_lo * state.num_attributes], 1, &beta, raw_g_lo_dotprod, 1 );
	
	}

	cout << "----------------\n";

	for (int i = 0 ; i < state.num_attributes; i++) {

		cout << g_lo_dotprod[i] << ",";

	}

	cout << "\n-------------\n";
//	t2 = CycleTimer::currentTicks();
//	cout << "SGEMV 2: " << t2-t1 << "\n";
//	t1 = t2;

	float x_hi_sq = g_x_sq[I_hi];
	float x_lo_sq = g_x_sq[I_lo];
		
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(g_hi_dotprod.begin(), g_lo_dotprod.begin(), g_x_sq.begin(), g_f.begin())),
   	                 thrust::make_zip_iterator(thrust::make_tuple(g_hi_dotprod.end(), g_lo_dotprod.end(), g_x_sq.end(),g_f.end())),
       	             update_functor(state.gamma, alpha_lo_old, alpha_hi_old, alpha_lo_new, alpha_hi_new, y_lo, y_hi, x_hi_sq, x_lo_sq));

	//prev_hi = I_hi;
	//prev_lo = I_lo;

//	t2 = CycleTimer::currentTicks();
//	cout << "UPDATE_FUNCTOR: " << t2-t1 << "\n";
//	t1 = t2;

/////////////////////////////////////////////////////////


//	t2 = CycleTimer::currentTicks();
//	cout << "Destroy: " << t2-t1 << "\n";
//	t1 = t2;
	return 0;
}

struct compare_mine
{
  __host__ __device__
  bool operator()(float lhs, float rhs)
  {
    return (lhs < rhs);
  }
};


int main(int argc, char *argv[]) {


    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");


	//Obtain the command line arguments
	parse_arguments(argc, argv);

	//input data attributes and labels
	std::vector<float> raw_x(state.num_train_data * state.num_attributes,0);// = new float[state.num_train_data * state.num_attributes];
	std::vector<int> raw_y(state.num_train_data,0);// = new int[state.num_train_data];

	//read data from input file
	cout << state.num_train_data << " " << state.num_attributes << " " << state.input_file_name << "\n";

	populate_data(raw_x, raw_y, state.num_train_data, state.num_attributes, state.input_file_name);
	cout << "Populated Data from input file\n";
	
	unsigned long long t1, t2, start;
	t1 = CycleTimer::currentTicks();
	start = CycleTimer::currentSeconds();
	
	thrust::host_vector<float> x (raw_x);
	thrust::host_vector<int> y (raw_y);


	//cout << "PRE COPY: 0\n";

	//Copy x and y to device
	thrust::device_vector<float> g_x (x.begin(), x.end());
	thrust::device_vector<int> g_y(y.begin(), y.end());
	
	thrust::device_vector<float> g_x_hi(state.num_attributes);
	thrust::device_vector<float> g_x_lo(state.num_attributes);
	
//	t2 = CycleTimer::currentTicks();

	//cout << "COPY: " << t2 - t1 << "\n";

//	t1 = t2;

	// Initialize f on device
	thrust::device_vector<float> g_f(state.num_train_data);
	thrust::transform(g_y.begin(), g_y.end(), g_f.begin(), thrust::negate<float>());

	//Initialize alpha on device
	thrust::device_vector<float> g_alpha(state.num_train_data, 0);
	
	//b (intercept), checks optimality condition for stopping
	float b_lo, b_hi;

	//check iteration number for stopping condition
	int num_iter = 0;
 
	thrust::host_vector<float> g_x_sq (state.num_train_data);

//	t2 = CycleTimer::currentTicks();
	//cout << "POST INIT, PRE G_X_SQ CALC: " << t2 - t1 << "\n";
//	t1 = t2;

	for( int i = 0; i < state.num_train_data; i++ )
	{
		g_x_sq[i] = thrust::inner_product(&g_x[i*state.num_attributes], &g_x[i*state.num_attributes] + state.num_attributes, &g_x[i*state.num_attributes], 0.0f);
	}

	t2 = CycleTimer::currentTicks();
	//cout << "G_X_SQ CALC: " << t2-t1 << "\n";
	t1 = t2;
	/*cout << "***g_x_sq***\n";
	for(int i=0; i<state.num_train_data; i++) {
		cout << g_x_sq[i] << '\n';
	}*/
	
	init_cuda_handles();
	t2 = CycleTimer::currentTicks();
	cout << "INIT_CUDA_HANDLES: " << t2-t1 << "\n";
	t1 = t2;
	
	lineCache = new myCache(10, state.num_attributes);

	t2 = CycleTimer::currentTicks();
	cout << "INIT CACHE: " << t2-t1 << "\n";
	t1 = t2;

	float* raw_g_x = thrust::raw_pointer_cast(&g_x[0]);
	//float* raw_g_f = thrust::raw_pointer_cast(&g_f[0]);

	thrust::device_vector<float>::iterator iter;
	//float* iter;
	do {

		cout << "Current iteration number: " << num_iter << "\n";
		
		//Set up I_set1 and I_set2
		thrust::device_vector<float> g_I_set1(state.num_train_data, 1000000000);
		thrust::device_vector<float> g_I_set2(state.num_train_data, -1000000000);
		
		thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(g_alpha.begin(), g_y.begin(), g_f.begin(), g_I_set1.begin(), g_I_set2.begin())),
    	                 thrust::make_zip_iterator(thrust::make_tuple(g_alpha.end(), g_y.end(), g_f.end(), g_I_set1.end(), g_I_set2.end())),
        	             arbitrary_functor(state.c));
	
	//	t2 = CycleTimer::currentTicks();
	//	cout << "I_SET CALC: " << t2 - t1 << "\n";
	//	t1 = t2;
		/*cout << "******g_I_set2******\n";
		for(int i=0; i<g_I_set2.size(); i++) {
			cout << g_I_set2[i] << "\n";
		}*/

		//get b_hi and b_low
		iter = thrust::max_element(g_I_set2.begin(), g_I_set2.end());//, compare_mine());

		int I_lo = iter - g_I_set2.begin();
		b_lo = *iter;

		//cout << "I_lo: \t" << I_lo << ", b_lo: \t" << b_lo << '\n';

		iter = thrust::min_element(g_I_set1.begin(), g_I_set1.end());

		int I_hi = iter - g_I_set1.begin();
		b_hi = *iter;

		//cout << "I_lo: \t" << I_lo << ", I_hi: \t" << I_hi << '\n';
		//cout << "b_lo: \t" << b_lo << ", b_hi: \t" << b_hi << '\n';

		int y_lo = y[I_lo];
		int y_hi = y[I_hi];
		
	//	t2 = CycleTimer::currentTicks();
	//	cout << "MAX_MIN CALC: " << t2 - t1 << "\n";
	//	t1 = t2;
		float eta = rbf_kernel(x,I_hi,I_hi) + rbf_kernel(x,I_lo,I_lo) - (2*rbf_kernel(x,I_lo,I_hi)) ;
		
	//	t2 = CycleTimer::currentTicks();
	//	cout << "ETA CALC: " << t2 - t1 << "\n";
	//	t1 = t2;
		//cout << "eta: " << eta << '\n';

		//obtain alpha_low and alpha_hi (old values)
		float alpha_lo_old = g_alpha[I_lo];
		float alpha_hi_old = g_alpha[I_hi];

		//update alpha_low and alpha_hi
		float s = y_lo*y_hi;
		float alpha_lo_new = alpha_lo_old + (y_lo*(b_hi - b_lo)/eta);
		float alpha_hi_new = alpha_hi_old + (s*(alpha_lo_old - alpha_lo_new));

		//clip new alpha values between 0 and C
		alpha_lo_new = clip_value(alpha_lo_new, 0.0, state.c);
		alpha_hi_new = clip_value(alpha_hi_new, 0.0, state.c);

		//cout << "alpha_lo_new: " << alpha_lo_new << '\n';
		//cout << "alpha_hi_new: " << alpha_hi_new << '\n';
		
		//store new alpha_1 and alpha_2 values
		g_alpha[I_lo] = alpha_lo_new;
		g_alpha[I_hi] = alpha_hi_new;

	//	t2 = CycleTimer::currentTicks();
	//	cout << "ALPHA UPDATE: " << t2-t1 << "\n";
	//	t1 = t2;
		//update f values
		update_f(g_f, raw_g_x, g_x_sq, I_lo, I_hi, y_lo, y_hi, alpha_lo_old, alpha_hi_old, alpha_lo_new, alpha_hi_new);

	//	t2 = CycleTimer::currentTicks();
	//	cout << "UPDATE_F: " << t2-t1 << "\n";
	//	t1 = t2;

		//Increment number of iterations to reach stopping condition
		num_iter++;

	//	cout << "--------------------------------\n";

	} while((b_lo > (b_hi +(2*state.epsilon))) && num_iter < state.max_iter);
	
	t2 = CycleTimer::currentSeconds();
	cout << "TOTAL TIME TAKEN in seconds: " << t2-start << "\n";

	if(b_lo > (b_hi + (2*state.epsilon))) {
		cout << "Could not converge in " << num_iter << " iterations. SVM training has been stopped\n";
	} else {
		cout << "Converged at iteration number: " << num_iter << "\n";
	}

	destroy_cuda_handles();

	//obtain final b intercept
	float b = (b_lo + b_hi)/2;
	cout << "b: " << b << "\n";

	//obtain training accuracy
	float train_accuracy = get_train_accuracy(x, y, g_alpha, b);
	cout << "Training accuracy: " << train_accuracy << "\n";

	//write model to file
	//write_out_model(x, y, alpha, b);

	//cout << "Training model has been saved to the file " << state.model_file_name << "\n";

	//clear training data
	//for(int i = 0 ; i < state.num_train_data; i++) {	
	//	delete [] x[i];
	//}

	//delete [] x;
	//delete [] y;

	return 0;
}

float get_train_accuracy(thrust::host_vector<float> &x, thrust::host_vector<int> &y, thrust::device_vector<float> &g_alpha, float b) {
	int num_correct = 0;

	thrust::host_vector<float> alpha = g_alpha; 
	float* raw_alpha = thrust::raw_pointer_cast(&alpha[0]);
	
	for(int i=0; i<state.num_train_data; i++) {
		//cout << "Iter: " << i << "\n";

		float dual = 0;

		for(int j=0; j<state.num_train_data; j++) {
			if(raw_alpha[j] != 0) {
				dual += y[j]*raw_alpha[j]*rbf_kernel(x,j,i);
			}
		}

		dual += b;

		int result = 1;
		if(dual < 0) {
			result = -1;
		}

		if(result == y[i]) {
			num_correct++;
		}
	}

	return ((float)num_correct/(state.num_train_data));
}

float clip_value(float num, float low, float high) {
	if(num < low) {
		return low;
	} else if(num > high) {
		return high;
	}

	return num;
}




void get_x(float* x, float* x_copy, int idx, int num_attributes) {
	int ctr = 0;

	int start_index = (idx*num_attributes);
	int end_index = start_index+num_attributes;

	for(int i = start_index; i < end_index; i++) {
		x_copy[ctr++] = x[i];
	}
}


float rbf_kernel(thrust::host_vector<float> &x, int i1, int i2){
	
	float* i2_copy = new float[state.num_attributes];

	float* raw_i1 = thrust::raw_pointer_cast(&x[i1*state.num_attributes]);
	float* raw_i2 = thrust::raw_pointer_cast(&x[i2*state.num_attributes]);

	get_x(raw_i2, i2_copy, 0, state.num_attributes);
	
	cblas_saxpy(state.num_attributes, -1, raw_i1, 1, i2_copy, 1); 

	//float norm = cblas_snrm2(state.num_attributes, i2_copy, 1);

	///float result = (float)exp(-1 *(float)state.gamma*norm*norm);

	float norm_sq = cblas_sdot(state.num_attributes, i2_copy, 1, i2_copy, 1);

	float result = (float)exp(-1 *(float)state.gamma*norm_sq);

	delete [] i2_copy;

	return result;
}
